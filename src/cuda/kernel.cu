#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include "interpolate.hh"
#include "Globals.h"
#include "kernel.hh"

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
 ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

__device__
double d_d_dtdm(double v,double e,double Y,double q){
  double v2=v*v;
  double v3=v2*v;
  double v4=v2*v2;
  double e2=e*e;
  double e4=e2*e2;
  double Y2=Y*Y;
  double q2=q*q;
  double eq=(2.*e4*(240. + v2*(-120. + v*(42.*(-27. + 4.*q2)*v + (-6567. + 1996.*q2)*v3 +
              48.*q*(8. + 77.*v2)*Y - 4.*q2*v*(90. + 1577.*v2)*Y2))) +
     e4*e2*(560. + v2*(-360. + 960.*q*v*Y + 8816.*q*v3*Y +
           v4*(-15565. + 24.*q2*(200. - 629.*Y2)) +
           v2*(-2742. + 80.*q2*(5. - 11.*Y2)))) -
     8.*e2*(-48. + v2*(8. - 64.*q*v*Y - 688.*q*v3*Y +
           2.*v2*(99. + 16.*q2*(-1. + 2.*Y2)) + v4*(1233. + 8.*q2*(-47. + 150.*Y2))))
       + 16.*(16. + v2*(24. + v2*(27.*(2. + 5.*v2) - 48.*q*v*Y +
              4.*q2*(2. - v2 + (-2. + 3.*v2)*Y2)))))/(256.*v4);
  return eq;
}

__device__
double d_d_dphidm(double v,double e,double Y,double q){
  double v2=v*v;
  double v3=v2*v;
  double e2=e*e;
  double e4=e2*e2;
  double Y2=Y*Y;
  double q2=q*q;
  double eq=(16. + 8.*(3. + e2)*v2 - 16.*q*v3*(-2. + (3. + e2)*Y) -
     8.*q*v3*v2*(-6. + 15.*Y + 3.*e4*Y + 2.*e2*(-4. + 7.*Y)) +
     2.*v2*v2*(27. + 3.*e4 + 2.*q2*(-1. + Y)*(1. + 7.*Y) + 2.*e2*(9. + q2*(1. + Y2))) +
     v3*v3*(5.*e4*e2 + e4*(45. + q2*(2. + 26.*Y2)) +
        e2*(135. + 4.*q2*(-19. + 5.*Y*(-7. + 9.*Y))) + 3.*(45. + 2.*q2*(-9. + Y*(-6. + 19.*Y)))))/(16.*v);
  return eq;
}

// ----- magnitude of azimuthal angular frequency for prograde/retrograde orbits -----
__device__
double d_OmegaPhi(double v, double e, double cosiota, double s, double M){

  double omegaphi;
  if(cosiota>0) omegaphi=d_d_dphidm(v,e,cosiota,s)/d_d_dtdm(v,e,cosiota,s)/M;
  else omegaphi=d_d_dphidm(v,e,-cosiota,-s)/d_d_dtdm(v,e,-cosiota,-s)/M;

  return omegaphi;

}

__device__
void d_cross(const double *u,const double *v,double *w){
  w[0] = u[1]*v[2]-u[2]*v[1];
  w[1] = u[2]*v[0]-u[0]*v[2];
  w[2] = u[0]*v[1]-u[1]*v[0];
}

__device__
double d_dot_product(const double *u,const double *v){
    return u[0]*v[0] + u[1]*v[1] + u[2]*v[2];
}

__device__
double d_vec_norm(const double *u){
    return sqrt(u[0]*u[0] + u[1]*u[1] + u[2]*u[2]);
}


__device__
void d_RotCoeff(double rot[],double iota,double theta_S,double phi_S,double theta_K,double phi_K,double alpha){


  double n[3];
  double L[3];
  double S[3];
  double nxL[3];
  double nxS[3];

  n[0] = sin(theta_S)*cos(phi_S);
  n[1] = sin(theta_S)*sin(phi_S);
  n[2] = cos(theta_S);
  S[0] = sin(theta_K)*cos(phi_K);
  S[1] = sin(theta_K)*sin(phi_K);
  S[2] = cos(theta_K);
  L[0] = cos(iota)*sin(theta_K)*cos(phi_K)+sin(iota)*(sin(alpha)*sin(phi_K)-cos(alpha)*cos(theta_K)*cos(phi_K));
  L[1] = cos(iota)*sin(theta_K)*sin(phi_K)-sin(iota)*(sin(alpha)*cos(phi_K)+cos(alpha)*cos(theta_K)*sin(phi_K));
  L[2] = cos(iota)*cos(theta_K)+sin(iota)*cos(alpha)*sin(theta_K);
  d_cross(n,L,nxL);
  d_cross(n,S,nxS);

  double norm=d_vec_norm(nxL)*d_vec_norm(nxS);
  double dot,cosrot,sinrot;
  //gsl_blas_ddot(nxL,nxS,&dot);
  dot = d_dot_product(nxL,nxS);
  cosrot=dot/norm;
  //gsl_blas_ddot(L,nxS,&dot);
  dot = d_dot_product(L,nxS);
  sinrot=dot;
  //gsl_blas_ddot(S,nxL,&dot);
  dot = d_dot_product(S,nxL);
  sinrot-=dot;
  sinrot/=norm;

  rot[0]=2.*cosrot*cosrot-1.;
  rot[1]=cosrot*sinrot;
  rot[2]=-rot[1];
  rot[3]=rot[0];
}

__device__
void find_index_and_xout(int *index, double *x_out, double *x_out2, double *x_out3, double dx, double x_new, double *x_old, int length){
    double x_trans;
    *index = (int)floor(x_new/dx);  // assumes first time is zero
    if (*index >= length - 1) *index = length - 2;
    x_trans = (x_new - x_old[*index]);

    *x_out = x_trans;
    *x_out2 = x_trans*x_trans;
    *x_out3 = x_trans*x_trans*x_trans;

    //printf("interp %d, %e %e, %e, %e, %e, %e\n", *index, dx, x_old[0], x_new, x_old[*index], x_old[*index+1], x_trans);

    /*# if __CUDA_ARCH__>=200
    if (x_new == 1.000100e+06)
        printf("interp %d, %e %e, %e, %e, %e, %e\n", *index, dx, x_old[0], x_new, x_old[*index], x_old[*index+1], x_trans);
    #endif //*/
}

__device__
double interpolate_array(InterpArrayContainer array_container, double x, double x2, double x3, int index, double x_new){
    double coeff_0 = array_container.array[index];
    double coeff_1 = array_container.coeff_1[index];
    double coeff_2 = array_container.coeff_2[index];
    double coeff_3 = array_container.coeff_3[index];
    double return_val = coeff_0 + coeff_1*x + coeff_2*x2 + coeff_3*x3;

    // printf("interp2 %d, %e, %e %e, %e, %e, %.18e, %.18e, %.18e, %.18e\n", index, return_val, x, x2, x3, x_new, coeff_0, coeff_1, coeff_2, coeff_3);

    /*# if __CUDA_ARCH__>=200
    if ((x_new <= 100.0))
        printf("interp2 %d, %e %e, %e, %e, %.18e, %.18e, %.18e, %.18e\n", index, x, x2, x3, x_new, coeff_0, coeff_1, coeff_2, coeff_3);
    #endif //*/

    return return_val;
}

__device__
double d_dtdm(double v,double e,double Y,double q){
  double v2=v*v;
  double v3=v2*v;
  double v4=v2*v2;
  double e2=e*e;
  double e4=e2*e2;
  double Y2=Y*Y;
  double q2=q*q;
  double eq=(2.*e4*(240. + v2*(-120. + v*(42.*(-27. + 4.*q2)*v + (-6567. + 1996.*q2)*v3 +
              48.*q*(8. + 77.*v2)*Y - 4.*q2*v*(90. + 1577.*v2)*Y2))) +
     e4*e2*(560. + v2*(-360. + 960.*q*v*Y + 8816.*q*v3*Y +
           v4*(-15565. + 24.*q2*(200. - 629.*Y2)) +
           v2*(-2742. + 80.*q2*(5. - 11.*Y2)))) -
     8.*e2*(-48. + v2*(8. - 64.*q*v*Y - 688.*q*v3*Y +
           2.*v2*(99. + 16.*q2*(-1. + 2.*Y2)) + v4*(1233. + 8.*q2*(-47. + 150.*Y2))))
       + 16.*(16. + v2*(24. + v2*(27.*(2. + 5.*v2) - 48.*q*v*Y +
              4.*q2*(2. - v2 + (-2. + 3.*v2)*Y2)))))/(256.*v4);
  return eq;
}

__device__
double d_drdm(double v,double e,double Y,double q){
  double v2=v*v;
  double v3=v2*v;
  double e2=e*e;
  double e4=e2*e2;
  double Y2=Y*Y;
  double q2=q*q;
  double eq=(16. + 8.*(-3. + e2)*v2 - 16.*(-3. + e2)*q*v3*Y +
     8.*(33. + 4.*e2 - 3.*e4)*q*v3*v2*Y +
     v3*v3*(-351. + 132.*q2 + e2*(-135. + 21.*e2 + 5.*e4 + 2.*(7. + e2)*q2) +
        2.*(-204. + 13.*e2*(-3. + e2))*q2*Y2) +
     2.*v2*v2*(-45. + 3.*e4 + 4.*q2*(1. - 4.*Y2) + 2.*e2*q2*(1. + Y2)))/(16.*v);
  return eq;
}

__device__
double d_dthetadm(double v,double e,double Y,double q){
  double v2=v*v;
  double v3=v2*v;
  double e2=e*e;
  double e4=e2*e2;
  double Y2=Y*Y;
  double q2=q*q;
  double eq=(16. + 8.*(3. + e2)*v2 - 16.*(3. + e2)*q*v3*Y -
     8.*(3. + e2)*(5. + 3.*e2)*q*v3*v2*Y +
     v3*v3*(135. - 54.*q2 + e2*(5.*(27. + 9.*e2 + e4) + 2.*(-38. + e2)*q2) +
        2.*(57. + 90.*e2 + 13.*e4)*q2*Y2) +
     2.*v2*v2*(27. + 3.*e4 + 2.*q2*(-1. + 7.*Y2) + 2.*e2*(9. + q2*(1. + Y2))))/(16.*v);
  return eq;
}

__device__
double d_dphidm(double v,double e,double Y,double q){
  double v2=v*v;
  double v3=v2*v;
  double e2=e*e;
  double e4=e2*e2;
  double Y2=Y*Y;
  double q2=q*q;
  double eq=(16. + 8.*(3. + e2)*v2 - 16.*q*v3*(-2. + (3. + e2)*Y) -
     8.*q*v3*v2*(-6. + 15.*Y + 3.*e4*Y + 2.*e2*(-4. + 7.*Y)) +
     2.*v2*v2*(27. + 3.*e4 + 2.*q2*(-1. + Y)*(1. + 7.*Y) + 2.*e2*(9. + q2*(1. + Y2))) +
     v3*v3*(5.*e4*e2 + e4*(45. + q2*(2. + 26.*Y2)) +
        e2*(135. + 4.*q2*(-19. + 5.*Y*(-7. + 9.*Y))) + 3.*(45. + 2.*q2*(-9. + Y*(-6. + 19.*Y)))))/(16.*v);
  return eq;
}

__global__
void produce_phasing(double *e_out, double *v_out, double *M_out, double *S_out, double *gimdot_out, double *nu_out, double *alpdot_out,
                    double *gim_out, double *Phi_out, double *alp_out,
                     double *tvec, InterpArrayContainer evec, InterpArrayContainer vvec, InterpArrayContainer Mvec, InterpArrayContainer Svec,
                            double lam,
                            int init_length,
                             double init_dt, double timestep, double t_clip, int run_length)
{

    int index;
    double x, x2, x3;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    double t = timestep*i;

    if (t > t_clip) return;

    double tm = timestep*(i-1);
    if (i==0) tm = t;

    double coslam=cos(lam);
    double sinlam=sin(lam);

    find_index_and_xout(&index, &x, &x2, &x3, init_dt, tm, tvec, init_length);

    double e=interpolate_array(evec, x, x2, x3, index, timestep*i); //evec.array[i];
    double v=interpolate_array(vvec, x, x2, x3, index, timestep*i); //vvec.array[i];
    double M=interpolate_array(Mvec, x, x2, x3, index, timestep*i); //Mvec.array[i];
    double S=interpolate_array(Svec, x, x2, x3, index, timestep*i); //Svec.array[i];

    double gimdotm=(d_dthetadm(v,e,coslam,S)-d_drdm(v,e,coslam,S))/d_dtdm(v,e,coslam,S)/M;
    double Phidotm=d_drdm(v,e,coslam,S)/d_dtdm(v,e,coslam,S)/M;
    double alpdotm=(d_dphidm(v,e,coslam,S)-d_dthetadm(v,e,coslam,S))/d_dtdm(v,e,coslam,S)/M;

    find_index_and_xout(&index, &x, &x2, &x3, init_dt, t, tvec, init_length);

    e=interpolate_array(evec, x, x2, x3, index, timestep*i); //evec.array[i];
    v=interpolate_array(vvec, x, x2, x3, index, timestep*i); //vvec.array[i];
    M=interpolate_array(Mvec, x, x2, x3, index, timestep*i); //Mvec.array[i];
    S=interpolate_array(Svec, x, x2, x3, index, timestep*i); //Svec.array[i];

    double gimdot=(d_dthetadm(v,e,coslam,S)-d_drdm(v,e,coslam,S))/d_dtdm(v,e,coslam,S)/M;
    double Phidot=d_drdm(v,e,coslam,S)/d_dtdm(v,e,coslam,S)/M;
    double alpdot=(d_dphidm(v,e,coslam,S)-d_dthetadm(v,e,coslam,S))/d_dtdm(v,e,coslam,S)/M;

    //double nu=Phidot/2./M_PI;

    e_out[i] = e;
    v_out[i] = v;
    M_out[i] = M;
    S_out[i] = S;
    gimdot_out[i] = gimdot;
    nu_out[i] = Phidot/2./M_PI;;
    alpdot_out[i] = alpdot;

    if (i >= run_length-1) return;
    gim_out[i+1] = (1.5*gimdot-.5*gimdotm)*timestep;
    Phi_out[i+1] = (1.5*Phidot-.5*Phidotm)*timestep;
    alp_out[i+1] = (1.5*alpdot-.5*alpdotm)*timestep;

    //double nu=Phidot/2./M_PI;

}

__global__ void prescan0(double *arr, double arr0, double *temp)
{
    *arr = arr0;
    *temp = 0.0;
}



__global__ void prescan1(double *g_idata, int n, double *temp, int num_sum_per_thread)
{
    int start_ind = (blockIdx.x*blockDim.x + threadIdx.x)*num_sum_per_thread;
    if (start_ind >= n) return;
    int end_ind = start_ind + num_sum_per_thread - 1;
    if (end_ind >= n) end_ind = n-1;

    for(int i=start_ind; i<end_ind; i++) g_idata[i+1]+=g_idata[i];
    int temp_ind = start_ind / num_sum_per_thread + 1;
    temp[temp_ind] = g_idata[end_ind];
}

__global__ void prescan2(int n, double *temp, int num_sum_per_thread)
{
    for(int i=0; i<n-1; i++) temp[i+1] += temp[i];
}

__global__ void prescan3(double *g_idata, int n, double *temp, int num_sum_per_thread)
{
    for(int i = blockIdx.x*blockDim.x + threadIdx.x;
        i<n;
        i += blockDim.x*gridDim.x){
            int temp_ind = i / num_sum_per_thread;
            g_idata[i] += temp[temp_ind];
        }
}

#define gpuErrchk_kern(ans) { gpuAssert_kern((ans), __FILE__, __LINE__); }
inline void gpuAssert_kern(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void cumsum(double *data, double phase0, int n){
    int num_sum_per_thread = 256;
    int NUM_THREADS = 256;
    int num_needed_threads = std::ceil(n/num_sum_per_thread);
    int num_blocks_prescan1 = std::ceil((num_needed_threads + 1 + NUM_THREADS -1)/NUM_THREADS);

    double *temp;
    gpuErrchk_kern(hipMalloc(&temp, (num_needed_threads+1)*sizeof(double)));


    prescan0<<<1,1>>>(data, phase0, temp);
    hipDeviceSynchronize();
    gpuErrchk_kern(hipGetLastError());

    prescan1<<<num_blocks_prescan1, NUM_THREADS>>>(data, n, temp, num_sum_per_thread);
    hipDeviceSynchronize();
    gpuErrchk_kern(hipGetLastError());

    prescan2<<<1,1>>>(num_needed_threads+1, temp, num_sum_per_thread);
    hipDeviceSynchronize();
    gpuErrchk_kern(hipGetLastError());

    int num_blocks_prescan3 = std::ceil((n + 1 + NUM_THREADS -1)/NUM_THREADS);
    prescan3<<<num_blocks_prescan3, NUM_THREADS>>>(data, n, temp, num_sum_per_thread);
    hipDeviceSynchronize();
    gpuErrchk_kern(hipGetLastError());

    gpuErrchk_kern(hipFree(temp));

}


__global__
void kernel_create_waveform(double *t, double *hI, double *hII,
                            double *tvec, double *evec, double *vvec, double *Mvec, double *Svec,
                            double *gimvec, double *Phivec, double *alpvec,
                            double *nuvec, double *gimdotvec, double lam,
                            double qS, double phiS, double qK, double phiK,
                            bool mich, int init_length, int vlength,int nmodes,
                            int i_plunge, int i_buffer, double zeta, double M_phys,
                            double init_dt, double timestep, int run_length){

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= vlength) return;
    if (i >= run_length) {
        hI[i] = 0.0;
        hII[i] = 0.0;
        return;
    }

  // ----------  // TODO: calculate this first section before gpu
  double coslam=cos(lam);
  double sinlam=sin(lam);
  double cosqS=cos(qS);
  double sinqS=sin(qS);
  double cosqK=cos(qK);
  double sinqK=sin(qK);
  double cosphiK=cos(phiK);
  double sinphiK=sin(phiK);
  double halfsqrt3=sqrt(3.)/2.;
  // ----- compute waveform from t_start to t_end -----
  //for(int i=0;i<vlength;i++){
  double time = timestep*i;
  t[i]= time;
  double t_plunge=i_plunge*timestep;
  double t_zero=t_plunge+timestep*i_buffer;

  if (time <=t_zero){

    hI[i]=0.;
    hII[i]=0.;

    double e=evec[i]; //evec.array[i];
    double v=vvec[i]; //vvec.array[i];
    double M=Mvec[i]; //Mvec.array[i];
    double S=Svec[i]; //Svec.array[i];
    double gim=gimvec[i]; //gimvec.array[i];
    double Phi=Phivec[i]; //Phivec.array[i];
    double alp=alpvec[i]; //alpvec.array[i];
    double nu=nuvec[i]; //nuvec.array[i];
    double gimdot=gimdotvec[i]; //gimdotvec.array[i];

    /*# if __CUDA_ARCH__>=200
    //if ((index >= 12000) && (index <= 12100))
        //printf("%d, %.18e, %.18e, %.18e, %.18e, %.18e, %.18e, %.18e, %.18e, %.18e, %.18e, %.18e\n", i, e, t[i], tvec[index], tvec[index+1], evec.array[index], evec.coeff_1[index], evec.coeff_2[index], evec.coeff_3[index], x, x2, x3);
        printf("%d, %.18e, %.18e, %.18e, %.18e\n", i, t[i], e, nu, gimdot);
    #endif //*/

    double cosalp=cos(alp);
    double sinalp=sin(alp);
    double cosqL=cosqK*coslam+sinqK*sinlam*cosalp;
    double sinqL=sqrt(1.-cosqL*cosqL);
    double phiLup=sinqK*sinphiK*coslam-cosphiK*sinlam*sinalp-cosqK*sinphiK*sinlam*cosalp;
    double phiLdown=sinqK*cosphiK*coslam+sinphiK*sinlam*sinalp-cosqK*cosphiK*sinlam*cosalp;
    double phiL=atan2(phiLup,phiLdown);
    double Ldotn=cosqL*cosqS+sinqL*sinqS*cos(phiL-phiS);
    double Ldotn2=Ldotn*Ldotn;
    double Sdotn=cosqK*cosqS+sinqK*sinqS*cos(phiK-phiS);
    double betaup=-Sdotn+coslam*Ldotn;
    double betadown=sinqS*sin(phiK-phiS)*sinlam*cosalp+(cosqK*Sdotn-cosqS)/sinqK*sinlam*sinalp;
    double beta=atan2(betaup,betadown);
    double gam=2.*(gim+beta);
    double cos2gam=cos(gam);
    double sin2gam=sin(gam);

    double orbphs,cosorbphs,sinorbphs,FplusI,FcrosI,FplusII,FcrosII;
    if(mich){

      orbphs=2.*M_PI*t[i]/year;
      cosorbphs=cos(orbphs-phiS);
      sinorbphs=sin(orbphs-phiS);

      double cosq=.5*cosqS-halfsqrt3*sinqS*cosorbphs;
      double phiw=orbphs+atan2(halfsqrt3*cosqS+.5*sinqS*cosorbphs,sinqS*sinorbphs);
      double psiup=.5*cosqK-halfsqrt3*sinqK*cos(orbphs-phiK)-cosq*(cosqK*cosqS+sinqK*sinqS*cos(phiK-phiS));
      double psidown=.5*sinqK*sinqS*sin(phiK-phiS)-halfsqrt3*cos(orbphs)*(cosqK*sinqS*sin(phiS)-cosqS*sinqK*sin(phiK))-halfsqrt3*sin(orbphs)*(cosqS*sinqK*cos(phiK)-cosqK*sinqS*cos(phiS));
      double psi=atan2(psiup,psidown);
      double cosq1=.5*(1.+cosq*cosq);
      double cos2phi=cos(2.*phiw);
      double sin2phi=sin(2.*phiw);
      double cos2psi=cos(2.*psi);
      double sin2psi=sin(2.*psi);

      FplusI=cosq1*cos2phi*cos2psi-cosq*sin2phi*sin2psi;
      FcrosI=cosq1*cos2phi*sin2psi+cosq*sin2phi*cos2psi;
      FplusII=cosq1*sin2phi*cos2psi+cosq*cos2phi*sin2psi;
      FcrosII=cosq1*sin2phi*sin2psi-cosq*cos2phi*cos2psi;
      if (i == 1000) printf("%d %e %e %e %e \n", i, FplusI, FplusII, FcrosI, FcrosII);
    }
    else{
      FplusI=1.;
      FcrosI=0.;
      FplusII=0.;
      FcrosII=1.;
    }

    double Amp=pow(d_OmegaPhi(v,e,coslam,S,M)*M_phys*SOLARMASSINSEC,2./3.)*zeta;

    // TODO: check making num modes to gridDim (then need to do reduction to get singular waveform)
    double fn,Doppler,nPhi;
    double ne, a, b, c, Aplus, Acros, Aplusold, Acrosold;
    double rot[4], J[5];
    for(int n=1;n<=nmodes;n++){

      if(mich){
        fn=n*nu+gimdot/M_PI;
        Doppler=2.*M_PI*fn*AUsec*sinqS*cosorbphs;
        nPhi=n*Phi+Doppler;
      }
      else nPhi=n*Phi;

      ne=n*e;
      if(n==1){
        J[0]=-1.0*j1(ne);
        J[1]=j0(ne);
        J[2]=j1(ne);
        J[3]=jn(2,ne);
        J[4]=jn(3,ne);
      }
      else{
          J[0]=jn(n-2, ne);
          J[1]=jn(n-1, ne);
          J[2]=jn(n, ne);
          J[3]=jn(n+1,ne);
          J[4]=jn(n+2,ne);
      }
      a=-n*Amp*(J[0]-2.*e*J[1]+2./n*J[2]+2.*e*J[3]-J[4])*cos(nPhi);
      b=-n*Amp*sqrt(1-e*e)*(J[0]-2.*J[2]+J[4])*sin(nPhi);
      c=2.*Amp*J[2]*cos(nPhi);
      Aplus=-(1.+Ldotn2)*(a*cos2gam-b*sin2gam)+c*(1-Ldotn2);
      Acros=2.*Ldotn*(b*cos2gam+a*sin2gam);

      // ----- rotate to NK wave frame -----
      Aplusold=Aplus;
      Acrosold=Acros;
      d_RotCoeff(rot,lam,qS,phiS,qK,phiK,alp);
      Aplus=Aplusold*rot[0]+Acrosold*rot[1];
      Acros=Aplusold*rot[2]+Acrosold*rot[3];
      // ----------

      double hnI,hnII;
      if(mich){
      	hnI=halfsqrt3*(FplusI*Aplus+FcrosI*Acros);
        hnII=halfsqrt3*(FplusII*Aplus+FcrosII*Acros);
      }
      else{
      	hnI=FplusI*Aplus+FcrosI*Acros;
        hnII=FplusII*Aplus+FcrosII*Acros;
      }

      hI[i]+=hnI;
      hII[i]+=hnII;

    }
  }

  if ((time>t_plunge) &&(i<vlength)){
    if(time<t_zero){
      hI[i]=hI[i]/(exp((t_plunge-t_zero)/(t[i]-t_plunge)+(t_plunge-t_zero)/(t[i]-t_zero))+1.);
      hII[i]=hII[i]/(exp((t_plunge-t_zero)/(t[i]-t_plunge)+(t_plunge-t_zero)/(t[i]-t_zero))+1.);
    }
    else{
      hI[i]=0.;
      hII[i]=0.;
  }
  // ----------

}
/*# if __CUDA_ARCH__>=200
  if (i == 1000)
      printf("%d, %.18e, %.18e\n", i, hI[i], hII[i]);
  #endif //*/

}

__global__
void likelihood_prep(hipDoubleComplex *template_channel1, hipDoubleComplex *template_channel2, double *noise_channel1_inv, double *noise_channel2_inv, int length){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= length) return;
    /*# if __CUDA_ARCH__>=200
      if (i == 1000)
          printf("%d, %.18e, %.18e, %.18e\n", i, hipCreal(template_channel1[i]), hipCimag(template_channel2[i]), noise_channel1_inv[i]);
      #endif //*/
    template_channel1[i] = hipCmul(template_channel1[i], make_hipDoubleComplex(noise_channel1_inv[i], 0.0));
    template_channel2[i] = hipCmul(template_channel2[i], make_hipDoubleComplex(noise_channel2_inv[i], 0.0));
}
