#include <assert.h>
#include <iostream>
#include "interpolate.hh"
#include <complex>
#include "hip/hip_complex.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


TrajectoryContainer * gpu_create_container(int max_length){
        TrajectoryContainer * trajectories;

        double *tvec, *evec, *Mvec, *Svec, *gimvec, *Phivec, *alpvec, *nuvec, *gimdotvec;
        double *evec_coeff_1, *Mvec_coeff_1, *Svec_coeff_1, *gimvec_coeff_1, *Phivec_coeff_1, *alpvec_coeff_1, *nuvec_coeff_1, *gimdotvec_coeff_1;
        double *evec_coeff_2, *Mvec_coeff_2, *Svec_coeff_2, *gimvec_coeff_2, *Phivec_coeff_2, *alpvec_coeff_2, *nuvec_coeff_2, *gimdotvec_coeff_2;
        double *evec_coeff_3, *Mvec_coeff_3, *Svec_coeff_3, *gimvec_coeff_3, *Phivec_coeff_3, *alpvec_coeff_3, *nuvec_coeff_3, *gimdotvec_coeff_3;

        gpuErrchk(hipMalloc(&trajectories, sizeof(TrajectoryContainer)));

            // waveform
            gpuErrchk(hipMalloc(&tvec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&evec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&Mvec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&Svec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&gimvec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&Phivec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&alpvec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&nuvec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&gimdotvec, max_length*sizeof(double)));

            gpuErrchk(hipMalloc(&evec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Mvec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Svec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimvec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Phivec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&alpvec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&nuvec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimdotvec_coeff_1, (max_length-1)*sizeof(double)));

            gpuErrchk(hipMalloc(&evec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Mvec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Svec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimvec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Phivec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&alpvec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&nuvec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimdotvec_coeff_2, (max_length-1)*sizeof(double)));

            gpuErrchk(hipMalloc(&evec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Mvec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Svec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimvec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Phivec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&alpvec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&nuvec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimdotvec_coeff_3, (max_length-1)*sizeof(double)));


            gpuErrchk(hipMemcpy(&(trajectories.tvec), &(tvec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.evec), &(evec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Mvec), &(Mvec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Svec), &(Svec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimvec), &(gimvec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Phivec), &(Phivec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.alpvec), &(alpvec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.nuvec), &(nuvec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimdotvec), &(gimdotvec), sizeof(double *), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(&(trajectories.evec_coeff_1), &(evec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Mvec_coeff_1), &(Mvec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Svec_coeff_1), &(Svec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimvec_coeff_1), &(gimvec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Phivec_coeff_1), &(Phivec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.alpvec_coeff_1), &(alpvec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.nuvec_coeff_1), &(nuvec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimdotvec_coeff_1), &(gimdotvec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(&(trajectories.evec_coeff_2), &(evec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Mvec_coeff_2), &(Mvec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Svec_coeff_2), &(Svec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimvec_coeff_2), &(gimvec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Phivec_coeff_2), &(Phivec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.alpvec_coeff_2), &(alpvec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.nuvec_coeff_2), &(nuvec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimdotvec_coeff_2), &(gimdotvec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(&(trajectories.evec_coeff_3), &(evec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Mvec_coeff_3), &(Mvec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Svec_coeff_3), &(Svec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimvec_coeff_3), &(gimvec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Phivec_coeff_3), &(Phivec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.alpvec_coeff_3), &(alpvec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.nuvec_coeff_3), &(nuvec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimdotvec_coeff_3), &(gimdotvec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));


        return trajectories;
}

void gpu_destroy_container(TrajectoryContainer * trajectories){
        gpuErrchk(hipFree(trajectories.tvec));
        gpuErrchk(hipFree(trajectories.evec));
        gpuErrchk(hipFree(trajectories.Mvec));
        gpuErrchk(hipFree(trajectories.Svec));
        gpuErrchk(hipFree(trajectories.gimvec));
        gpuErrchk(hipFree(trajectories.Phivec));
        gpuErrchk(hipFree(trajectories.alpvec));
        gpuErrchk(hipFree(trajectories.nuvec));
        gpuErrchk(hipFree(trajectories.gimdotvec));

        gpuErrchk(hipFree(trajectories.evec_coeff_1));
        gpuErrchk(hipFree(trajectories.Mvec_coeff_1));
        gpuErrchk(hipFree(trajectories.Svec_coeff_1));
        gpuErrchk(hipFree(trajectories.gimvec_coeff_1));
        gpuErrchk(hipFree(trajectories.Phivec_coeff_1));
        gpuErrchk(hipFree(trajectories.alpvec_coeff_1));
        gpuErrchk(hipFree(trajectories.nuvec_coeff_1));
        gpuErrchk(hipFree(trajectories.gimdotvec_coeff_1));

        gpuErrchk(hipFree(trajectories.evec_coeff_2));
        gpuErrchk(hipFree(trajectories.Mvec_coeff_2));
        gpuErrchk(hipFree(trajectories.Svec_coeff_2));
        gpuErrchk(hipFree(trajectories.gimvec_coeff_2));
        gpuErrchk(hipFree(trajectories.Phivec_coeff_2));
        gpuErrchk(hipFree(trajectories.alpvec_coeff_2));
        gpuErrchk(hipFree(trajectories.nuvec_coeff_2));
        gpuErrchk(hipFree(trajectories.gimdotvec_coeff_2));

        gpuErrchk(hipFree(trajectories.evec_coeff_3));
        gpuErrchk(hipFree(trajectories.Mvec_coeff_3));
        gpuErrchk(hipFree(trajectories.Svec_coeff_3));
        gpuErrchk(hipFree(trajectories.gimvec_coeff_3));
        gpuErrchk(hipFree(trajectories.Phivec_coeff_3));
        gpuErrchk(hipFree(trajectories.alpvec_coeff_3));
        gpuErrchk(hipFree(trajectories.nuvec_coeff_3));
        gpuErrchk(hipFree(trajectories.gimdotvec_coeff_3));


    gpuErrchk(hipFree(trajectories));
}



TrajectoryContainer * cpu_create_container(int max_length){
        TrajectoryContainer * trajectories;

        double *tvec, *evec, *Mvec, *Svec, *gimvec, *Phivec, *alpvec, *nuvec, *gimdotvec;
        double *evec_coeff_1, *Mvec_coeff_1, *Svec_coeff_1, *gimvec_coeff_1, *Phivec_coeff_1, *alpvec_coeff_1, *nuvec_coeff_1, *gimdotvec_coeff_1;
        double *evec_coeff_2, *Mvec_coeff_2, *Svec_coeff_2, *gimvec_coeff_2, *Phivec_coeff_2, *alpvec_coeff_2, *nuvec_coeff_2, *gimdotvec_coeff_2;
        double *evec_coeff_3, *Mvec_coeff_3, *Svec_coeff_3, *gimvec_coeff_3, *Phivec_coeff_3, *alpvec_coeff_3, *nuvec_coeff_3, *gimdotvec_coeff_3;

        gpuErrchk(hipMalloc(&trajectories, sizeof(TrajectoryContainer)));

            // waveform
            gpuErrchk(hipMalloc(&tvec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&evec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&Mvec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&Svec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&gimvec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&Phivec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&alpvec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&nuvec, max_length*sizeof(double)));
            gpuErrchk(hipMalloc(&gimdotvec, max_length*sizeof(double)));

            gpuErrchk(hipMalloc(&evec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Mvec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Svec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimvec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Phivec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&alpvec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&nuvec_coeff_1, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimdotvec_coeff_1, (max_length-1)*sizeof(double)));

            gpuErrchk(hipMalloc(&evec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Mvec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Svec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimvec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Phivec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&alpvec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&nuvec_coeff_2, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimdotvec_coeff_2, (max_length-1)*sizeof(double)));

            gpuErrchk(hipMalloc(&evec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Mvec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Svec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimvec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&Phivec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&alpvec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&nuvec_coeff_3, (max_length-1)*sizeof(double)));
            gpuErrchk(hipMalloc(&gimdotvec_coeff_3, (max_length-1)*sizeof(double)));


            gpuErrchk(hipMemcpy(&(trajectories.tvec), &(tvec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.evec), &(evec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Mvec), &(Mvec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Svec), &(Svec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimvec), &(gimvec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Phivec), &(Phivec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.alpvec), &(alpvec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.nuvec), &(nuvec), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimdotvec), &(gimdotvec), sizeof(double *), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(&(trajectories.evec_coeff_1), &(evec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Mvec_coeff_1), &(Mvec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Svec_coeff_1), &(Svec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimvec_coeff_1), &(gimvec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Phivec_coeff_1), &(Phivec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.alpvec_coeff_1), &(alpvec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.nuvec_coeff_1), &(nuvec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimdotvec_coeff_1), &(gimdotvec_coeff_1), sizeof(double *), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(&(trajectories.evec_coeff_2), &(evec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Mvec_coeff_2), &(Mvec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Svec_coeff_2), &(Svec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimvec_coeff_2), &(gimvec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Phivec_coeff_2), &(Phivec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.alpvec_coeff_2), &(alpvec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.nuvec_coeff_2), &(nuvec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimdotvec_coeff_2), &(gimdotvec_coeff_2), sizeof(double *), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(&(trajectories.evec_coeff_3), &(evec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Mvec_coeff_3), &(Mvec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Svec_coeff_3), &(Svec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimvec_coeff_3), &(gimvec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.Phivec_coeff_3), &(Phivec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.alpvec_coeff_3), &(alpvec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.nuvec_coeff_3), &(nuvec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&(trajectories.gimdotvec_coeff_3), &(gimdotvec_coeff_3), sizeof(double *), hipMemcpyHostToDevice));


        return trajectories;
}

void cpu_destroy_container(TrajectoryContainer * trajectories){
        gpuErrchk(hipFree(trajectories.tvec));
        gpuErrchk(hipFree(trajectories.evec));
        gpuErrchk(hipFree(trajectories.Mvec));
        gpuErrchk(hipFree(trajectories.Svec));
        gpuErrchk(hipFree(trajectories.gimvec));
        gpuErrchk(hipFree(trajectories.Phivec));
        gpuErrchk(hipFree(trajectories.alpvec));
        gpuErrchk(hipFree(trajectories.nuvec));
        gpuErrchk(hipFree(trajectories.gimdotvec));

        gpuErrchk(hipFree(trajectories.evec_coeff_1));
        gpuErrchk(hipFree(trajectories.Mvec_coeff_1));
        gpuErrchk(hipFree(trajectories.Svec_coeff_1));
        gpuErrchk(hipFree(trajectories.gimvec_coeff_1));
        gpuErrchk(hipFree(trajectories.Phivec_coeff_1));
        gpuErrchk(hipFree(trajectories.alpvec_coeff_1));
        gpuErrchk(hipFree(trajectories.nuvec_coeff_1));
        gpuErrchk(hipFree(trajectories.gimdotvec_coeff_1));

        gpuErrchk(hipFree(trajectories.evec_coeff_2));
        gpuErrchk(hipFree(trajectories.Mvec_coeff_2));
        gpuErrchk(hipFree(trajectories.Svec_coeff_2));
        gpuErrchk(hipFree(trajectories.gimvec_coeff_2));
        gpuErrchk(hipFree(trajectories.Phivec_coeff_2));
        gpuErrchk(hipFree(trajectories.alpvec_coeff_2));
        gpuErrchk(hipFree(trajectories.nuvec_coeff_2));
        gpuErrchk(hipFree(trajectories.gimdotvec_coeff_2));

        gpuErrchk(hipFree(trajectories.evec_coeff_3));
        gpuErrchk(hipFree(trajectories.Mvec_coeff_3));
        gpuErrchk(hipFree(trajectories.Svec_coeff_3));
        gpuErrchk(hipFree(trajectories.gimvec_coeff_3));
        gpuErrchk(hipFree(trajectories.Phivec_coeff_3));
        gpuErrchk(hipFree(trajectories.alpvec_coeff_3));
        gpuErrchk(hipFree(trajectories.nuvec_coeff_3));
        gpuErrchk(hipFree(trajectories.gimdotvec_coeff_3));


    gpuErrchk(hipFree(trajectories));
}
